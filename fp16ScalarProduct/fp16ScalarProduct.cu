#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "hip/hip_fp16.h"
#include "hip/hip_runtime_api.h"

#include <cstdio>
#include <cstdlib>
#include <ctime>

#define NUM_OF_BLOCKS 80
#define NUM_OF_THREADS 256

__forceinline__ __device__ void reduceInShared_intrinsics(half2 *const v) {
  if (threadIdx.x < 64)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 64]);
  __syncthreads();
  if (threadIdx.x < 32)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 32]);
  __syncthreads();
  if (threadIdx.x < 16)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 16]);
  __syncthreads();
  if (threadIdx.x < 8)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 8]);
  __syncthreads();
  if (threadIdx.x < 4)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 4]);
  __syncthreads();
  if (threadIdx.x < 2)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 2]);
  __syncthreads();
  if (threadIdx.x < 1)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 1]);
  __syncthreads();
}

__forceinline__ __device__ void reduceInShared_native(half2 *const v) {
  if (threadIdx.x < 64) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 64];
  __syncthreads();
  if (threadIdx.x < 32) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 32];
  __syncthreads();
  if (threadIdx.x < 16) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x < 8) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x < 4) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x < 2) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x < 1) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 1];
  __syncthreads();
}

__device__ unsigned int get_smid(void) {
    unsigned int ret;
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
    return ret;
}

__global__ void scalarProductKernel_intrinsics(half2 const *const a,
                                               half2 const *const b,
                                               float *const results,
                                               size_t const size, int ovsize, int *SM_ids) {
 int smid = get_smid();

   for (int j=0; j < ovsize; j++){
        if (smid == SM_ids[j]){

  const int stride = ovsize * blockDim.x;
  __shared__ half2 shArray[NUM_OF_THREADS];

  shArray[threadIdx.x] = __float2half2_rn(0.f);
  half2 value = __float2half2_rn(0.f);

  for (int i = threadIdx.x + blockDim.x + j; i < size; i += stride) {
    value = __hfma2(a[i], b[i], value);
  }

  shArray[threadIdx.x] = value;
  __syncthreads();
  reduceInShared_intrinsics(shArray);

  if (threadIdx.x == 0) {
    half2 result = shArray[0];
    float f_result = __low2float(result) + __high2float(result);
    results[j] = f_result;
  }
}
   }
                                               }

__global__ void scalarProductKernel_native(half2 const *const a,
                                           half2 const *const b,
                                           float *const results,
                                           size_t const size, int ovsize, int *SM_ids) {

 int smid = get_smid();

   for (int j=0; j < ovsize; j++){
        if (smid == SM_ids[j]){

  const int stride = blockDim.x * ovsize;
  __shared__ half2 shArray[NUM_OF_THREADS];

  half2 value(0.f, 0.f);
  shArray[threadIdx.x] = value;

  for (int i = threadIdx.x + blockDim.x + j; i < size; i += stride) {
    value = a[i] * b[i] + value;
  }

  shArray[threadIdx.x] = value;
  __syncthreads();
  reduceInShared_native(shArray);

  if (threadIdx.x == 0) {
    half2 result = shArray[0];
    float f_result = (float)result.y + (float)result.x;
    results[j] = f_result;
  }
}
}
}

void generateInput(half2 *a, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    half2 temp;
    temp.x = static_cast<float>(rand() % 4);
    temp.y = static_cast<float>(rand() % 2);
    a[i] = temp;
  }
}

int main(int argc, char *argv[]) {
  srand((unsigned int)time(NULL));
  size_t size = 5000000;

  half2 *vec[2];
  half2 *devVec[2];

  float *results;
  float *devResults;

    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    // Check if command line arguments were provided
    int GPCs[6] = {0};
    int num_GPCs = 0;

    for (int i = 1; i < argc; i++) {
    if (strcmp(argv[i], "-gpc") == 0 && i + 1 < argc) {
            // Split the next argument by commas
            char* gpc_arg = argv[i + 1];
            char* gpc_token = strtok(gpc_arg, ",");
            while (gpc_token != NULL) {
                // Convert the token to an integer and use it as a GPC
                int gpc = atoi(gpc_token);
                GPCs[num_GPCs++] = gpc;
                // Get the next token
                gpc_token = strtok(NULL, ",");
            }
        }
    }

      // Define the GPC arrays
    unsigned int GPC_arrays[6][14] = {
        {0, 12, 24, 36, 48, 60, 70, 1, 13, 25, 37, 49, 61, 71},
        {2, 14, 26, 38, 50, 62, 72, 3, 15, 27, 39, 51, 63, 73},
        {4, 16, 28, 40, 52, 64, 74, 5, 17, 29, 41, 53, 65, 75},
        {6, 18, 30, 42, 54, 66, 76, 7, 19, 31, 43, 55, 67, 77},
        {8, 20, 32, 44, 56, 68, 9, 21, 33, 45, 57, 69},
        {10, 22, 34, 46, 58, 78, 11, 23, 35, 47, 59, 79}
    };

    // Compute the overall size and create the overall array
    int overall_size = 0;
    for (int i = 0; i < num_GPCs; i++) {
        overall_size += (GPCs[i] < 4) ? 14 : 12;
    }
    int *hSM_ids = (int *)malloc(sizeof(int) * overall_size);
    int *dSM_ids;
    hipMalloc((void**)&dSM_ids, sizeof(int) * overall_size);

    // Fill the overall array
    int current_position = 0;
    for (int i = 0; i < num_GPCs; i++) {
        int GPC_size = (GPCs[i] < 4) ? 14 : 12;
        memcpy(&hSM_ids[current_position], GPC_arrays[GPCs[i]], sizeof(int) * GPC_size);
        current_position += GPC_size;
    }

    //printf("%i\n", overall_size);

    // Copy the host array to the device
    hipMemcpy(dSM_ids, hSM_ids, sizeof(int) * overall_size, hipMemcpyHostToDevice);

  int devID = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t devProp;
  checkCudaErrors(hipGetDeviceProperties(&devProp, devID));

/*   if (devProp.major < 5 || (devProp.major == 5 && devProp.minor < 3)) {
    printf(
        "ERROR: fp16ScalarProduct requires GPU devices with compute SM 5.3 or "
        "higher.\n");
    return EXIT_WAIVED;
  } */

  for (int i = 0; i < 2; ++i) {
    checkCudaErrors(hipHostMalloc((void **)&vec[i], size * sizeof *vec[i]));
    checkCudaErrors(hipMalloc((void **)&devVec[i], size * sizeof *devVec[i]));
  }

  checkCudaErrors(
      hipHostMalloc((void **)&results, NUM_OF_BLOCKS * sizeof *results));
  checkCudaErrors(
      hipMalloc((void **)&devResults, NUM_OF_BLOCKS * sizeof *devResults));

  for (int i = 0; i < 2; ++i) {
    generateInput(vec[i], size);
    checkCudaErrors(hipMemcpy(devVec[i], vec[i], size * sizeof *vec[i],
                               hipMemcpyHostToDevice));
  }


    long flops = 2 * size - 1;
    int repeat = 100;

  hipEventRecord(beg);
    for (int i=0; i<repeat; i++){
  scalarProductKernel_native<<<NUM_OF_BLOCKS, NUM_OF_THREADS>>>(
      devVec[0], devVec[1], devResults, size, overall_size, dSM_ids);
    }
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.; // Convert to seconds

/*         printf(
        "%7.1f\n", //GFLOPS
        (repeat * flops * 1e-9) / elapsed_time);
    fflush(stdout); */

/*   checkCudaErrors(hipMemcpy(results, devResults,
                             NUM_OF_BLOCKS * sizeof *results,
                             hipMemcpyDeviceToHost));

  float result_native = 0;
  for (int i = 0; i < NUM_OF_BLOCKS; ++i) {
    result_native += results[i];
  }
  printf("Result native operators\t: %f \n", result_native); */

  hipEventRecord(beg);
    for (int i=0; i<repeat; i++){
  scalarProductKernel_intrinsics<<<NUM_OF_BLOCKS, NUM_OF_THREADS>>>(
      devVec[0], devVec[1], devResults, size, overall_size, dSM_ids);
    }
      hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.; // Convert to seconds

        printf(
        "%7.1f\n", //GFLOPS
        (repeat * flops * 1e-9) / elapsed_time);
    fflush(stdout);

/*   checkCudaErrors(hipMemcpy(results, devResults,
                             NUM_OF_BLOCKS * sizeof *results,
                             hipMemcpyDeviceToHost));

  float result_intrinsics = 0;
  for (int i = 0; i < NUM_OF_BLOCKS; ++i) {
    result_intrinsics += results[i];
  }
  printf("Result intrinsics\t: %f \n", result_intrinsics);

  printf("&&&& fp16ScalarProduct %s\n",
         (fabs(result_intrinsics - result_native) < 0.00001) ? "PASSED"
                                                             : "FAILED"); */

  for (int i = 0; i < 2; ++i) {
    checkCudaErrors(hipFree(devVec[i]));
    checkCudaErrors(hipHostFree(vec[i]));
  }

  checkCudaErrors(hipFree(devResults));
  checkCudaErrors(hipHostFree(results));

  return EXIT_SUCCESS;
}
