/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <stdlib.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

__device__ unsigned int get_smid(void) {
    unsigned int ret;
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
    return ret;
}


__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements, int size, int *SM_ids)
{
    int smid = get_smid();

    for (int j=0; j < size; j++){
        if (smid == SM_ids[j]){
            for (int i = threadIdx.x+blockDim.x*j; i < numElements; i += size*blockDim.x)
            {
                C[i] = A[i] + B[i];
            }
        }
    }
}

/**
 * Host main routine
 */
int main(int argc, char *argv[])
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    // Print the vector length to be used, and compute its size
    int numElements = 5000000;
    int threadsPerBlock = 256;
    int blocksPerGrid = 80;
    int coreMax = 1;

    // Check if command line arguments were provided
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-size") == 0 && i + 1 < argc) {
            // Convert the next argument to an integer and use it as the size
            numElements = atoi(argv[i + 1]);
        }
        else if (strcmp(argv[i], "-threads") == 0 && i + 1 < argc) {
            // Convert the next argument to an integer and use it as the number of threads per block
            threadsPerBlock = atoi(argv[i + 1]);
        }
        else if (strcmp(argv[i], "-blocks") == 0 && i + 1 < argc) {
            // Convert the next argument to an integer and use it as the number of blocks per grid
            blocksPerGrid = atoi(argv[i + 1]);
        }
        else if (strcmp(argv[i], "-coremax") == 0 && i + 1 < argc) {
            // Convert the next argument to an integer and use it as the maximum core number
            coreMax = atoi(argv[i + 1]);
        }
    }

    // Allocate the hSM_ids array and fill it with numbers from 0 to coreMax
    int *hSM_ids = (int *)malloc(sizeof(int) * (coreMax));
    for (int i = 0; i < coreMax; i++) {
        hSM_ids[i] = i;
    }

    // Allocate the dSM_ids array and copy the hSM_ids array to the device
    int *dSM_ids;
    hipMalloc((void**)&dSM_ids, sizeof(int) * (coreMax ));
    hipMemcpy(dSM_ids, hSM_ids, sizeof(int) * (coreMax ), hipMemcpyHostToDevice);

    size_t size = numElements * sizeof(float);
    //printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    //printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel

    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    long flops = 2 * numElements;
    int repeat = 100;

    hipEventRecord(beg);
    for (int i=0; i<repeat; i++){
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements, coreMax, dSM_ids);
    }
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.; // Convert to seconds
    err = hipGetLastError();

    printf(
        "%7.1f\n", //GFLOPS
        (repeat * flops * 1e-9) / elapsed_time);
    fflush(stdout);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    //printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    //printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //printf("Done\n");
    return 0;
}

